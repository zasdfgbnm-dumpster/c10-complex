#include <c10/test/util/complex_test_common.h>

__managed__ bool pass;

TEST(NonStaticHostTests, all) {
  constructors::test_thrust_conversion();
  assignment::test_assign_thrust();
  io::test_io();
  test_std::test_values();
}

int main() {
  NonStaticHostTests_all();
}