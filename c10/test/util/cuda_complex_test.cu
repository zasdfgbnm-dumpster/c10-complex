#include "hip/hip_runtime.h"
#include <c10/test/util/complex_test_common.h>

TEST(HostTests, all) {
  run_all_host_tests();
}

__global__ void test_thrust_kernel() {
  // thrust conversion
  {
  constexpr float num1 = float(1.23);
  constexpr float num2 = float(4.56);
  assert(c10::complex<float>(thrust::complex<float>(num1, num2)).real() == num1);
  assert(c10::complex<float>(thrust::complex<float>(num1, num2)).imag() == num2);
  }
  {
  constexpr double num1 = double(1.23);
  constexpr double num2 = double(4.56);
  assert(c10::complex<double>(thrust::complex<double>(num1, num2)).real() == num1);
  assert(c10::complex<double>(thrust::complex<double>(num1, num2)).imag() == num2);
  }
  // thrust assignment
  auto tup = assignment::one_two_thrust();
  assert(std::get<c10::complex<double>>(tup).real() == double(1));
  assert(std::get<c10::complex<double>>(tup).imag() == double(2));
  assert(std::get<c10::complex<float>>(tup).real() == float(1));
  assert(std::get<c10::complex<float>>(tup).imag() == float(2));
}

__global__ void test_std_functions_kernel() {
}

TEST(DeviceTests, ThrustConversion) {
  hipDeviceSynchronize();
  test_thrust_kernel<<<1, 1>>>();
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
}

TEST(DeviceTests, StdFunctions) {
  hipDeviceSynchronize();
  test_std_functions_kernel<<<1, 1>>>();
  hipDeviceSynchronize();
  ASSERT_EQ(hipGetLastError(), hipSuccess);
}

int main() {
  HostTests_all();
  DeviceTests_ThrustConversion();
  DeviceTests_StdFunctions();
}